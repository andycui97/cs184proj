#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include "main.h"
#include "random.h"


using namespace optix;

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
};

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

static __device__ __inline__ float2 concentric_sample_disk(const float2& uOffset)
{
    // ConcentricSampleDisk
    // a) Map uniform random number to [-1,1]^2
    //float2 uOffset = 2.f * u - make_float2(1.f, 1.f);

    // b) Handle degeneracy at origin
    if (uOffset.x == 0 && uOffset.y == 0)
        return make_float2(0.f, 0.f);

    // c) Apply concentric mapping to point
    float theta;
    float r;
    if(abs(uOffset.x) > abs(uOffset.y))
    {
        r = uOffset.x;
        theta = M_PI_4f * (uOffset.y / uOffset.x);
    }
    else
    {
        r = uOffset.y;
        theta = M_PI_2f - M_PI_4f * (uOffset.x / uOffset.y);
    }


    return r * make_float2(cos(theta), sin(theta));
}

static __device__ __inline__ float2 get_disk_sample(const float2& u,
    const float maxwidth,
    const float maxheight,
    const uchar2& index)
{

    float w = (-(1 / 2) + ((index.x + 0.5) / maxwidth));
        
    float h = ((1 / 2) - ((index.y + 0.5) / maxheight));

    // a) Map uniform random number to [-1,1]^2
    float2 uOffset = 2.f * u - make_float2(1.f, 1.f);

    float2 image_point = make_float2(w, h);

    image_point += uOffset;

    return concentric_sample_disk(image_point);
}


// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );




//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );

rtBuffer<float4, 2>              output_buffer;
rtBuffer<ParallelogramLight>     lights;


RT_PROGRAM void pathtrace_camera()
{
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
    do 
    {
        //
        // Sample pixel using jittering
        //
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
        float2 d = pixel + jitter*jitter_scale;
        float3 ray_origin = eye;
        float3 ray_direction = normalize(d.x*U + d.y*V + W);

        // Initialze per-ray data
        PerRayData_pathtrace prd;
        prd.result = make_float3(0.f);
        prd.attenuation = make_float3(1.f);
        prd.countEmitted = true;
        prd.done = false;
        prd.seed = seed;
        prd.depth = 0;

        // Each iteration is a segment of the ray path.  The closest hit will
        // return new segments to be traced here.
        for(;;)
        {
            Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            rtTrace(top_object, ray, prd);

            if(prd.done)
            {
                // We have hit the background or a luminaire
                prd.result += prd.radiance * prd.attenuation;
                break;
            }

            // Russian roulette termination 
            if(prd.depth >= rr_begin_depth)
            {
                float pcont = fmaxf(prd.attenuation);
                if(rnd(prd.seed) >= pcont)
                    break;
                prd.attenuation /= pcont;
            }

            prd.depth++;
            prd.result += prd.radiance * prd.attenuation;

            // Update ray data for the next path segment
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }

        result += prd.result;
        seed = prd.seed;
    } while (--samples_per_pixel);

    //
    // Update the output buffer
    //
    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

    if (frame_number > 1)
    {
        float a = 1.0f / (float)frame_number;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4( lerp( old_color, pixel_color, a ), 1.0f );
    }
    else
    {
        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
    }
}

rtDeclareVariable(float, f_length, , );
rtDeclareVariable(float, lens_rad, , );

RT_PROGRAM void thin_lens_camera()
{


    // If lens_radius is 0, treat as pinhole
    if(lens_rad < .00001f)
    {
        pathtrace_camera();
        return;
    }

    // Get ray direction of eye to image plane in the same way as before
    size_t2 screen = output_buffer.size();
    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 d = (make_float2(launch_index)) * inv_screen - 1.f;

    float3 init_ray_origin = eye;
    float3 init_ray_direction = normalize(d.x*U + d.y*V + W);

    float ft = f_length / dot(init_ray_direction,normalize(W));
    float3 pFocus = init_ray_origin + init_ray_direction * ft;

    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
    do 
    {
        //
        // Sample pixel using jittering
        //
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;

        float2 sample_point = make_float2(rnd(seed), rnd(seed));

        float2 disc_point = get_disk_sample(sample_point,
            sqrt_num_samples,
            sqrt_num_samples,
            make_uchar2(x,y));

        float2 p_lens = lens_rad * disc_point;

        // 2) Compute point on plane of focus
        float3 ray_origin = init_ray_origin + p_lens.x * U + p_lens.y * V;
        float3 ray_direction = normalize(pFocus - ray_origin);
        // printf("pre o %f %f %f\n", init_ray_origin.x, init_ray_origin.y, init_ray_origin.z);
        // printf("post o %f %f %f\n", ray_origin.x, ray_origin.y, ray_origin.z);
        // printf("pre d %f %f %f\n", init_ray_direction.x, init_ray_direction.y, init_ray_direction.z);
        // printf("post d %f %f %f\n", ray_direction.x, ray_direction.y, ray_direction.z);


        // float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
        // float2 d = pixel + jitter*jitter_scale;
        // float3 ray_origin = eye;
        // float3 ray_direction = normalize(-d.x*U - d.y*V + W);

        // Initialze per-ray data
        PerRayData_pathtrace prd;
        prd.result = make_float3(0.f);
        prd.attenuation = make_float3(1.f);
        prd.countEmitted = true;
        prd.done = false;
        prd.seed = seed;
        prd.depth = 0;

        // Each iteration is a segment of the ray path.  The closest hit will
        // return new segments to be traced here.
        for(;;)
        {
            Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            rtTrace(top_object, ray, prd);

            if(prd.done)
            {
                // We have hit the background or a luminaire
                prd.result += prd.radiance * prd.attenuation;
                break;
            }

            // Russian roulette termination 
            if(prd.depth >= rr_begin_depth)
            {
                float pcont = fmaxf(prd.attenuation);
                if(rnd(prd.seed) >= pcont)
                    break;
                prd.attenuation /= pcont;
            }

            prd.depth++;
            prd.result += prd.radiance * prd.attenuation;

            // Update ray data for the next path segment
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }

        result += prd.result;
        seed = prd.seed;
    } while (--samples_per_pixel);

    //
    // Update the output buffer
    //
    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

    if (frame_number > 1)
    {
        float a = 1.0f / (float)frame_number;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4( lerp( old_color, pixel_color, a ), 1.0f );
    }
    else
    {
        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
    }
}



//-----------------------------------------------------------------------------
//
//  Emissive surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
    current_prd.done = true;
}


//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,     diffuse_color, , );
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );


RT_PROGRAM void diffuse()
{
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;

    //
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //
    current_prd.origin = hitpoint;

    float z1=rnd(current_prd.seed);
    float z2=rnd(current_prd.seed);
    float3 p;
    cosine_sample_hemisphere(z1, z2, p);
    optix::Onb onb( ffnormal );
    onb.inverse_transform( p );
    current_prd.direction = p;

    // NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
    // with cosine density.
    current_prd.attenuation = current_prd.attenuation * diffuse_color;
    current_prd.countEmitted = false;

    //
    // Next event estimation (compute direct lighting).
    //
    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for(int i = 0; i < num_lights; ++i)
    {
        // Choose random point on light
        ParallelogramLight light = lights[i];
        const float z1 = rnd(current_prd.seed);
        const float z2 = rnd(current_prd.seed);
        const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = length(light_pos - hitpoint);
        const float3 L     = normalize(light_pos - hitpoint);
        const float  nDl   = dot( ffnormal, L );
        const float  LnDl  = dot( light.normal, L );

        // cast shadow ray
        if ( nDl > 0.0f && LnDl > 0.0f )
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
                const float A = length(cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
                result += light.emission * weight;
            }
        }
    }

    current_prd.radiance = result;
}


RT_PROGRAM void mirror()
{
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;

    //
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //
    current_prd.origin = hitpoint;
    current_prd.direction = reflect(ray.direction, ffnormal );


    // NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
    // with cosine density.
    current_prd.attenuation = current_prd.attenuation * diffuse_color;
    current_prd.countEmitted = true;

    //
    // Next event estimation (compute direct lighting).
    //
    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for(int i = 0; i < num_lights; ++i)
    {
        // Choose random point on light
        ParallelogramLight light = lights[i];
        const float z1 = rnd(current_prd.seed);
        const float z2 = rnd(current_prd.seed);
        const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = length(light_pos - hitpoint);
        const float3 L     = normalize(light_pos - hitpoint);
        const float  nDl   = dot( ffnormal, L );
        const float  LnDl  = dot( light.normal, L );

    }

    current_prd.radiance = result;

}

/*
Glass instructions:
If there is total internal reflection
    assign the reflection of wo to *wi
    set the *pdf to 1
    return reflectance / abs_cos_theta(*wi)
Else
    Compute Schlick's reflection coefficient RR
    If coin_flip(R)
        assign the reflection of wo to *wi
        set the *pdf to R
        return R * reflectance / abs_cos_theta(*wi)
    Else
        assign the refraction of wo to *wi
        set the *pdf to 1-R
        return (1-R) * transmittance / abs_cos_theta(*wi) / eta^2 (where eta^2 is the same as in the refraction function)
*/

rtDeclareVariable(float,        refraction_index, , );

RT_PROGRAM void glass()
{
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;

    //
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //
    current_prd.origin = hitpoint;

    //EDIT HERE TO IMPLEMENT GLASS

    float3 i = ray.direction;                                            // incident direction
    float3 t;                                                            // transmission direction
    float3 r;                                                            // reflection direction
    float3 n = world_shading_normal;

    if ( refract(t, i, n, refraction_index) )
    {
        current_prd.direction = t;
    } 
    else
    {
        current_prd.direction = reflect(ray.direction, ffnormal );
    }


    // NOTE: For diffuse, f/pdf = 1 since we are perfectly importance sampling lambertian
    // with cosine density.
    current_prd.attenuation = current_prd.attenuation * diffuse_color;
    current_prd.countEmitted = true;

    //
    // Next event estimation (compute direct lighting).
    //
    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for(int i = 0; i < num_lights; ++i)
    {
        // Choose random point on light
        ParallelogramLight light = lights[i];
        const float z1 = rnd(current_prd.seed);
        const float z2 = rnd(current_prd.seed);
        const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = length(light_pos - hitpoint);
        const float3 L     = normalize(light_pos - hitpoint);
        const float  nDl   = dot( ffnormal, L );
        const float  LnDl  = dot( light.normal, L );

        // cast shadow ray
        if ( nDl > 0.0f && LnDl > 0.0f )
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
                const float A = length(cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
                result += light.emission * weight;
            }
        }
    }

    current_prd.radiance = result;


}


rtBuffer<float4> planes;
RT_PROGRAM void chull_intersect(int primIdx)
{
    int n = planes.size();
    float t0 = -exp(800.0); //FLT_MIN
    float t1 = exp(800.0); //FLT_MAX
    float3 t0_normal = make_float3(0);
    float3 t1_normal = make_float3(0);
    for(int i = 0; i < n && t0 < t1; ++i ) {
    float4 plane = planes[i];
    float3 n = make_float3(plane);
    float  d = plane.w;

    float denom = dot(n, ray.direction);
    float t = -(d + dot(n, ray.origin))/denom;
    if( denom < 0){
        // enter
        if(t > t0){
        t0 = t;
        t0_normal = n;
        }
    } else {
        //exit
        if(t < t1){
        t1 = t;
        t1_normal = n;
        }
    }
    }
    if(t0 > t1)
    return;

// intersection program continued from above
    if(rtPotentialIntersection( t0 )){
    shading_normal = geometric_normal = t0_normal;
    rtReportIntersection(0);
    } else if(rtPotentialIntersection( t1 )){
    shading_normal = geometric_normal = t1_normal;
    rtReportIntersection(0);
    }
}




RT_PROGRAM void chull_bounds (int primIdx, float result[6])
{
   optiX::Aabb* aabb = (Aabb*) result;
//    aabb->m_min = chull_bbmin;
//    aabb->m_max = chull_bbmax;
}




//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}


//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, bg_color, , );

RT_PROGRAM void miss()
{
    current_prd.radiance = bg_color;
    current_prd.done = true;
}


rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void envmap_miss()
{
  float theta = atan2f( ray.direction.x, ray.direction.z );
  float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
  float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
  float v     = 0.5f * ( 1.0f + sin(phi) );
  current_prd.radiance = make_float3( tex2D(envmap, u, v) );
  current_prd.done = true;

}
  

RT_PROGRAM void tex()
{

    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;

    //
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //
    current_prd.origin = hitpoint;
    current_prd.direction = ffnormal ;


  float theta = atan2f( current_prd.direction.x, current_prd.direction.z );
  float phi   = M_PIf * 0.5f -  acosf( current_prd.direction.y );
  float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
  float v     = 0.5f * ( 1.0f + sin(phi) );
  current_prd.radiance = make_float3( tex2D(envmap, u, v) );
  current_prd.done = true;

}
